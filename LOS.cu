#include "hip/hip_runtime.h"
﻿#include "gdal_priv.h"
#include "cpl_conv.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <fstream>
#include "hip/hip_runtime.h"
#include ""
#include <chrono>

#define DEG2RAD(x) ((x) * 3.14159265f / 180.0f)
#define MAX_RAYS 360  // Safe with heap allocation

using namespace std;

// ---------------- CUDA Kernel ----------------
struct HillHit {
    int x, y;
    bool valid;
};

__global__ void computeLOS(
    const float* tile, int width, int height,
    int cx, int cy, float centerElev,
    const float* angles, int numRays,
    HillHit* hits, int maxSteps)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= numRays) return;

    float angle = DEG2RAD(angles[idx]);
    float dx = cosf(angle);
    float dy = sinf(angle);

    for (int step = 1; step < maxSteps; ++step) {
        int x = roundf(cx + dx * step);
        int y = roundf(cy + dy * step);

        if (x < 0 || x >= width || y < 0 || y >= height) break;

        float elev = tile[y * width + x];

        if (elev > centerElev) {
            hits[idx] = { x, y, true };
            break;
        }
    }
}

// ---------------- Utilities ----------------

void checkCuda(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        cerr << "CUDA Error: " << msg << ": " << hipGetErrorString(err) << endl;
        exit(1);
    }
}

void PixelToWorld(double gt[6], int px, int py, double& x, double& y) {
    x = gt[0] + px * gt[1] + py * gt[2];
    y = gt[3] + px * gt[4] + py * gt[5];
}

void prepareAngles(float* angles, int numRays, float step) {
    for (int i = 0; i < numRays; ++i) {
        angles[i] = i * step;
    }
}

void writeKMLWithPolygon(
    const vector<HillHit>& hits,
    int width, int height,
    double gt[6], int cx, int cy,
    const string& filename)
{
    ofstream kml(filename);
    kml << "<?xml version=\"1.0\" encoding=\"UTF-8\"?>\n";
    kml << "<kml xmlns=\"http://www.opengis.net/kml/2.2\">\n<Document>\n";

    // Style for hill points
    kml << "  <Style id=\"hillStyle\">\n"
        << "    <IconStyle><color>ff0000ff</color><scale>1.0</scale>\n"
        << "    <Icon><href>http://maps.google.com/mapfiles/kml/shapes/placemark_circle.png</href></Icon>\n"
        << "    </IconStyle>\n  </Style>\n";

    // Style for center point
    kml << "  <Style id=\"centerStyle\">\n"
        << "    <IconStyle><color>ff00ff00</color><scale>1.2</scale>\n"
        << "    <Icon><href>http://maps.google.com/mapfiles/kml/shapes/placemark_circle.png</href></Icon>\n"
        << "    </IconStyle>\n  </Style>\n";

    // Hill point markers
    for (const auto& h : hits) {
        if (!h.valid) continue;
        double lon, lat;
        PixelToWorld(gt, h.x, h.y, lon, lat);
        kml << "<Placemark><styleUrl>#hillStyle</styleUrl><Point><coordinates>"
            << lon << "," << lat << ",0</coordinates></Point></Placemark>\n";
    }

    // Center marker
    double centerLon, centerLat;
    PixelToWorld(gt, cx, cy, centerLon, centerLat);
    kml << "<Placemark><styleUrl>#centerStyle</styleUrl><Point><coordinates>"
        << centerLon << "," << centerLat << ",0</coordinates></Point></Placemark>\n";

    // Polygon around hill hits
    kml << "<Placemark><name>LOS Polygon</name>\n"
        << "  <Style><LineStyle><color>ff0000ff</color><width>2</width></LineStyle>\n"
        << "  <PolyStyle><color>7f0000ff</color></PolyStyle></Style>\n"
        << "  <Polygon><outerBoundaryIs><LinearRing><coordinates>\n";

    for (const auto& h : hits) {
        if (!h.valid) continue;
        double lon, lat;
        PixelToWorld(gt, h.x, h.y, lon, lat);
        kml << lon << "," << lat << ",0\n";
    }

    // Close polygon
    for (const auto& h : hits) {
        if (h.valid) {
            double lon, lat;
            PixelToWorld(gt, h.x, h.y, lon, lat);
            kml << lon << "," << lat << ",0\n";
            break;
        }
    }

    kml << "</coordinates></LinearRing></outerBoundaryIs></Polygon></Placemark>\n";
    kml << "</Document>\n</kml>\n";
}

// ---------------- Main ----------------

int main() {

    auto start = std::chrono::high_resolution_clock::now();


    GDALAllRegister();
    string tifPath = "E:/tiles/all_merged.tif";         // Place your own path here

    GDALDataset* ds = (GDALDataset*)GDALOpen(tifPath.c_str(), GA_ReadOnly);
    if (!ds) {
        cerr << "Failed to open file!\n";
        return 1;
    }

    int width = ds->GetRasterXSize();
    int height = ds->GetRasterYSize();
    GDALRasterBand* band = ds->GetRasterBand(1);
    double gt[6];
    ds->GetGeoTransform(gt);

    vector<float> tile(width * height);
    band->RasterIO(GF_Read, 0, 0, width, height, tile.data(), width, height, GDT_Float32, 0, 0);

    int cx = width / 2;
    int cy = height / 2;
    float centerElev = tile[cy * width + cx];

    // Allocate angles on heap to avoid stack overflow
    vector<float> angles(MAX_RAYS);
    prepareAngles(angles.data(), MAX_RAYS, 360.0f / MAX_RAYS);

    int maxSteps = (int)sqrtf((float)(width * width + height * height));

    // Allocate CUDA memory
    float* d_tile;
    float* d_angles;
    HillHit* d_hits;
    checkCuda(hipMalloc(&d_tile, width * height * sizeof(float)), "alloc tile");
    checkCuda(hipMalloc(&d_angles, MAX_RAYS * sizeof(float)), "alloc angles");
    checkCuda(hipMalloc(&d_hits, MAX_RAYS * sizeof(HillHit)), "alloc hits");

    checkCuda(hipMemcpy(d_tile, tile.data(), width * height * sizeof(float), hipMemcpyHostToDevice), "copy tile");
    checkCuda(hipMemcpy(d_angles, angles.data(), MAX_RAYS * sizeof(float), hipMemcpyHostToDevice), "copy angles");
    checkCuda(hipMemset(d_hits, 0, MAX_RAYS * sizeof(HillHit)), "clear hits");

    computeLOS << <(MAX_RAYS + 255) / 256, 256 >> > (
        d_tile, width, height, cx, cy, centerElev,
        d_angles, MAX_RAYS, d_hits, maxSteps
        );
    checkCuda(hipDeviceSynchronize(), "kernel");

    vector<HillHit> hits(MAX_RAYS);
    checkCuda(hipMemcpy(hits.data(), d_hits, MAX_RAYS * sizeof(HillHit), hipMemcpyDeviceToHost), "copy hits");

    writeKMLWithPolygon(hits, width, height, gt, cx, cy, "E:/tiles/LOS_1000p.kml");     // Place your own path here
    cout << "Exported KML to E:/tiles/output_LOS_polygon.kml\n";                        // Place your own path here

    GDALClose(ds);
    hipFree(d_tile);
    hipFree(d_angles);
    hipFree(d_hits);

    // Stop timer
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate duration
    std::chrono::duration<double> duration = end - start;

    std::cout << "Execution time: " << duration.count() << " seconds" << std::endl;

    return 0;
}
